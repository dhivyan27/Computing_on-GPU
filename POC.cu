
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void vector_add(float *A, float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

__global__ void scalar_multiply(float *C, float *D, float scalar, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        D[i] = C[i] * scalar;
    }
}

__global__ void fused_operation(float *A, float *B, float *D, float scalar, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        float C = A[i] + B[i];
        D[i] = C * scalar;
    }
}

int main() {
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float scalarValue = 2.0;

    // Memory allocation and initialization code
    float *h_A = new float[numElements];
    float *h_B = new float[numElements];
    for (int i = 0; i < numElements; i++) {
        h_A[i] = rand() / static_cast<float>(RAND_MAX);
        h_B[i] = rand() / static_cast<float>(RAND_MAX);
    }

    float *d_A, *d_B, *d_C, *d_D;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMalloc((void**)&d_D, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    const int threadsPerBlock = 256;  
    const int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Non-fused version timing
    float milliseconds1 = 0;
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1);
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    scalar_multiply<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_D, scalarValue, numElements);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    hipEventElapsedTime(&milliseconds1, start1, stop1);

    // Fused version timing
    float milliseconds2 = 0;
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    fused_operation<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, scalarValue, numElements);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds2, start2, stop2);

    // Allocate memory for host results
    float *h_Result_NonFused = new float[numElements];
    float *h_Result_Fused = new float[numElements];

    // Copy results from device to host for non-fused operation
    hipMemcpy(h_Result_NonFused, d_D, size, hipMemcpyDeviceToHost);

    // Execute fused operation
    fused_operation<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, scalarValue, numElements);

    // Copy results from device to host for fused operation
    hipMemcpy(h_Result_Fused, d_D, size, hipMemcpyDeviceToHost);

    // Compare and print results
    bool resultsMatch = true;
    for (int i = 0; i < numElements; i++) {
        printf("Element %d: Non-Fused = %f, Fused = %f\n", i, h_Result_NonFused[i], h_Result_Fused[i]);
        if (h_Result_NonFused[i] != h_Result_Fused[i]) {
            resultsMatch = false;
        }
    }

    if (resultsMatch) {
        printf("Results match!\n");
    } else {
        printf("Results differ!\n");
    }

    printf("Time taken by non-fused version: %f ms\n", milliseconds1);
    printf("Time taken by fused version: %f ms\n", milliseconds2);




    // Cleanup code
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    return 0;
}
